/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005 
#define TILE_H 32
#define TILE_W 32
#define FILTER_RADIUS 32
#define FILTER_LENGTH 	(2 * FILTER_RADIUS  + 1)
#define MAX_SIZE 4096   /* Maximum possible size of submatrix */
#define STREAMS 4


__device__ __constant__ double d_Filter[FILTER_LENGTH];

#define cudaCheckError() {                                                                       \
        hipError_t error=hipGetLastError();                                                        \
        if(error!=hipSuccess) {                                                                     \
            printf("ERROR IN CUDA %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(error));        \
            hipDeviceReset(); \
            exit(EXIT_FAILURE);                                                                  \
        }                                                                                        \
    }

/* Separable convolution kernel in regards to ROWS.
 * Arguments:
 * 		- d_Dst: output array, buffered to COLUMN kernel
 * 		- d_Src: source array
 * 		- padding_right: array that contains the right-most padding elements of original array
 * 		- padding_left: same as padding_right, but for the left-most padding elements 
 * 		- imageW/H: the width/height of the image matrix
 * 		- option: [0,1,2] --
 *						   |-> 0: the kernel only uses the padding_left variable to calculate the last column
 *						   |-> 1: the kernel only uses the padding_right variable to calculate the 1st column
 *						   |-> 2: the kernel uses both paddings to calculate all elements in between
 */
__global__ void convolutionRowGPU(double *d_Dst, double *d_Src,double *padding_right,double *padding_left,int imageW, int imageH,int option) {
    
    int x,y,k,d,P_x,x0;
   
    __shared__ double image[TILE_H * (TILE_W+FILTER_RADIUS*2)];  
    
    x =  blockIdx.x*blockDim.x + threadIdx.x;
    y =  blockIdx.y*blockDim.y + threadIdx.y;
    
    P_x = x - FILTER_RADIUS;
    
    /* Divergent code iff FILTER_RADIUS < 32 BUT can be customized depending on threads/block */
    if(P_x < 0) {
        if(option == 0 || option == 2) {
            image[threadIdx.x + threadIdx.y*(TILE_W+FILTER_RADIUS*2)] = padding_left[y*FILTER_RADIUS + x];
        }
        else{
            image[threadIdx.x + threadIdx.y*(TILE_W+FILTER_RADIUS*2)] = 0;
        }
        image[threadIdx.x + blockDim.x + threadIdx.y*(TILE_W+FILTER_RADIUS*2)] = d_Src[y*imageW + x + blockDim.x - FILTER_RADIUS];
        image[threadIdx.x + blockDim.x + FILTER_RADIUS + threadIdx.y*(TILE_W+FILTER_RADIUS*2)] = d_Src[y*imageW + x + blockDim.x];
    }
    else{
        if(threadIdx.x < FILTER_RADIUS) {
            image[threadIdx.x + threadIdx.y*(TILE_W+FILTER_RADIUS*2)] = d_Src[y*imageW + x - FILTER_RADIUS];
            image[threadIdx.x + blockDim.x + threadIdx.y*(TILE_W+FILTER_RADIUS*2)] = d_Src[y*imageW + x + blockDim.x - FILTER_RADIUS];
            image[threadIdx.x + blockDim.x + FILTER_RADIUS + threadIdx.y*(TILE_W+FILTER_RADIUS*2)] = d_Src[y*imageW + x + blockDim.x];
        }
        else{
            image[threadIdx.x + threadIdx.y*(TILE_W+FILTER_RADIUS*2)] = d_Src[y*imageW + x - FILTER_RADIUS];
        }
            
    }

    P_x = x + FILTER_RADIUS;
    
    if(P_x > imageW - 1) {
        if(option == 1 || option == 2) {
            image[threadIdx.x + 2*FILTER_RADIUS +  threadIdx.y*(TILE_W+FILTER_RADIUS*2)] = padding_right[y*FILTER_RADIUS + P_x-imageW];
        }
        else{
           image[threadIdx.x + 2*FILTER_RADIUS +  threadIdx.y*(TILE_W+FILTER_RADIUS*2)] = 0;
        }
        
        
        image[threadIdx.x + FILTER_RADIUS +  threadIdx.y*(TILE_W+FILTER_RADIUS*2)] = d_Src[y*imageW + x];
    }
    
    
    
    __syncthreads();
    

    
    double sum = 0;
    
    x0 = threadIdx.x + FILTER_RADIUS ;
    for(k = -FILTER_RADIUS; k <= FILTER_RADIUS; k++) {
        d = x0 + k;
        
        sum += image[threadIdx.y*(TILE_W+FILTER_RADIUS*2) + d] * d_Filter[FILTER_RADIUS - k];
    }
        

    d_Dst[y*imageW + x] = sum;
    
}

/* Separable convolution kernel in regards to COLUMNS.
 * Arguments:
 * 		- d_Dst: final output array
 * 		- d_Src: source array, buffered from d_Dst of ROW kernel
 * 		- padding_up: array that contains the up-most padding elements of original array
 * 		- padding_down: same as padding_up, but for the down-most padding elements 
 * 		- imageW/H: the width/height of the image matrix
 * 		- option: [0,1,2] --
 *						   |-> 0: the kernel only uses the padding_up variable to calculate the last row
 *						   |-> 1: the kernel only uses the padding_down variable to calculate the 1st row
 *						   |-> 2: the kernel uses both paddings to calculate all elements in between
 */
__global__ void convolutionColumnGPU(double *d_Dst, double *d_Src,double *padding_up,double *padding_down,int imageW, int imageH,int option) {
    
      
    int x,y,k,d,P_y,y0;
   
    __shared__ double image[TILE_W * (TILE_H+FILTER_RADIUS*2)];  
    
    x =  blockIdx.x*blockDim.x + threadIdx.x;
    y =  blockIdx.y*blockDim.y + threadIdx.y;
    
    
    P_y = y - FILTER_RADIUS;

	/* Same as row kernel, divergent iff FILTER_RADIUS < 32 */    
    if(P_y < 0) {
        if(option == 0 || option == 2) {
            image[threadIdx.y*(TILE_W) + threadIdx.x] = padding_up[y*imageW + x];
        }
        else{
            image[threadIdx.y*(TILE_W) + threadIdx.x] = 0;
        }
        image[(threadIdx.y + blockDim.y)*TILE_W + threadIdx.x] = d_Src[(y + blockDim.y - FILTER_RADIUS)*imageW + x ];
        image[(threadIdx.y + blockDim.y + FILTER_RADIUS)*TILE_W + threadIdx.x] = d_Src[(y + blockDim.y)*imageW + x ];
    }
    else{
        if(threadIdx.y < FILTER_RADIUS){
            
            if(y + blockDim.y > imageH - 1) {
                image[threadIdx.y*(TILE_W) + threadIdx.x] = d_Src[(y- FILTER_RADIUS)*imageW + x];
            }
            else {
                image[threadIdx.y*TILE_W + threadIdx.x] = d_Src[(y- FILTER_RADIUS)*imageW + x];
                image[(threadIdx.y + blockDim.y)*TILE_W + threadIdx.x] = d_Src[(y + blockDim.y - FILTER_RADIUS)*imageW + x ];
                image[(threadIdx.y + blockDim.y + FILTER_RADIUS)*TILE_W + threadIdx.x] = d_Src[(y + blockDim.y)*imageW + x ];
            }
                
        }
        else{
            image[threadIdx.y*(TILE_W) + threadIdx.x] = d_Src[(y- FILTER_RADIUS)*imageW + x];
        }
    }

    P_y = y + FILTER_RADIUS;
    
    if(P_y > imageH - 1) {
        if(option == 1 || option == 2) {
            image[(threadIdx.y+2*FILTER_RADIUS)*TILE_W + threadIdx.x] = padding_down[(P_y-imageH)*imageW + x];
        }
        else{
            image[(threadIdx.y+2*FILTER_RADIUS)*TILE_W + threadIdx.x] = 0;
        }
        
        image[(threadIdx.y + FILTER_RADIUS)*TILE_W + threadIdx.x] = d_Src[y*imageW + x];
    }
        
    __syncthreads();
    

 
    double sum = 0;
    
    y0 = threadIdx.y + FILTER_RADIUS ;
    for(k = -FILTER_RADIUS; k <= FILTER_RADIUS; k++) {
        d = y0 + k;
        
        sum += image[d*TILE_W + threadIdx.x] * d_Filter[FILTER_RADIUS - k];
        
    }
    d_Dst[y*imageW + x] = sum;
    
}

 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////


void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, 
                       int imageW, int imageH) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -FILTER_RADIUS; k <= FILTER_RADIUS; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[FILTER_RADIUS - k];
        }     
      }

      
      //printf("ROW X:%d Y:%d SUM:%f\n\n",x,y,sum);
      h_Dst[y * imageW + x] = sum;
    }
  }
        
}



////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -FILTER_RADIUS; k <= FILTER_RADIUS; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[FILTER_RADIUS - k];
        }   
      }
      //printf("COL X:%d Y:%d SUM:%f\n\n",x,y,sum);
      h_Dst[y * imageW + x] = sum;
    }
  }
    
}


//


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,
    *h_Input,
    *h_input_new,
    *h_filters_row_start,
    *h_filters_row_end,
    *h_filters_col_up,
    *h_filters_col_down,
    *h_Buffer_new,
    *d_Input_stream0,
    *d_Input_stream1,
    *d_Input_stream2,
    *d_Input_stream3,
    *d_Buffer_stream0,
    *d_Buffer_stream1,
    *d_Buffer_stream2,
    *d_Buffer_stream3,
    *d_OutputGPU_stream0,
    *d_OutputGPU_stream1,
    *d_OutputGPU_stream2,
    *d_OutputGPU_stream3,
    *d_Filter_col_up_stream0,
    *d_Filter_col_up_stream1,
    *d_Filter_col_up_stream2,
    *d_Filter_col_up_stream3,
    *d_Filter_col_down_steam0,
    *d_Filter_col_down_steam1,
    *d_Filter_col_down_steam2,
    *d_Filter_col_down_steam3,
    *d_Filter_row_left_stream0,
    *d_Filter_row_left_stream1,
    *d_Filter_row_left_stream2,
    *d_Filter_row_left_stream3,
    *d_Filter_row_right_stream0,
    *d_Filter_row_right_stream1,
    *d_Filter_row_right_stream2,
    *d_Filter_row_right_stream3,
    *h_OutputGPU,
    *h_OutputGPU_new;
#ifdef _HOST
    *h_Buffer,
    *h_OutputCPU,
    *h_Buffer_debug,
#endif


    int imageW;
    int imageH;
    unsigned int i;
    int number_of_blocks;
    int array_W;
    int array_H;
    int x;
    int y;
    int j;
    
#ifdef _HOST
    double timing;
    clock_t start;
    clock_t end;
#endif
    hipStream_t stream0, stream1 , stream2, stream3;

    

 
    
    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;
    
    /* number_of_blocks <= 256 strictly */
    printf("Enter number of blocks should be a power of 2 (this number will be squared for actual calculations, e.g. 4->real blocks = 16): ");
    scanf("%d", &number_of_blocks);
    imageH = imageW;
    

    dim3 threads(TILE_H,TILE_W);
    dim3 blocks (number_of_blocks,number_of_blocks);
    
    array_H = TILE_H*number_of_blocks;
    array_W = TILE_W*number_of_blocks;
    
    
    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays and device array...\n");
    
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
#ifdef _HOST
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer_debug = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
#endif
    h_OutputGPU = (double *)malloc(imageW * imageH * sizeof(double));   
    
    /* Allocating pinned memory for row elements */
    hipHostMalloc((void**)&h_input_new, imageW * imageH * sizeof(double), hipHostMallocDefault);
    cudaCheckError();
    hipHostMalloc((void**)&h_Buffer_new, imageW * imageH * sizeof(double), hipHostMallocDefault);
    cudaCheckError();
    hipHostMalloc((void**)&h_filters_row_start,imageH/array_H * (imageW/array_W-1)* array_H * FILTER_RADIUS *sizeof(double), hipHostMallocDefault);
    cudaCheckError();
    hipHostMalloc((void**)& h_filters_row_end, imageH/array_H * (imageW/array_W-1)* array_H * FILTER_RADIUS *sizeof(double), hipHostMallocDefault);
    cudaCheckError();

    
    if (h_Filter==NULL || h_Input == NULL){
        printf("Something went wrong wille malloc in CPU\n");
        exit(EXIT_FAILURE);
    }
    
    printf("Memory allocation for host arrays: COMPLETED \n");
    
    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
    }
    
    for(i=0; i < imageH/array_H; i++) {
        for(j=0; j < imageW/array_W; j++) {
            for(y=0; y < array_H; y++) {
                for(x=0; x < array_W;x++) {
                    
                    h_input_new[y*array_W + x + (i*imageH/array_H + j)*array_H*array_W]= h_Input[(y+i*array_H)*imageW + x + j*array_W];
                    
                    
                    if(j != 0 && x < FILTER_RADIUS) {
                        h_filters_row_start[x + y*FILTER_RADIUS +(i*(imageH/array_H-1)+j-1)*FILTER_RADIUS*array_H] = h_input_new[y*array_W + x + (i*imageH/array_H + j)*array_H*array_W];
                    }
                    
                    if(j != imageW/array_W -1 && x+FILTER_RADIUS > array_W-1){
                        h_filters_row_end[x- array_W+FILTER_RADIUS + y*FILTER_RADIUS +(i*(imageH/array_H-1)+j)*FILTER_RADIUS*array_H] = h_input_new[y*array_W + x + (i*imageH/array_H + j)*array_H*array_W];

                    }
                }
            }
        }
    }
    
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    
    hipMemcpyToSymbol(HIP_SYMBOL(d_Filter),h_Filter,FILTER_LENGTH * sizeof(double),0,hipMemcpyHostToDevice);
    
    cudaCheckError();
    
    hipMallocManaged((void**)&d_Input_stream0,array_H * array_W * sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Input_stream1,array_H * array_W * sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Input_stream2,array_H * array_W * sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Input_stream3,array_H * array_W * sizeof(double));
    cudaCheckError();
    
    hipMallocManaged((void**)&d_Buffer_stream0,array_H * array_W * sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Buffer_stream1,array_H * array_W * sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Buffer_stream2,array_H * array_W * sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Buffer_stream3,array_H * array_W * sizeof(double));
    cudaCheckError();
    
    hipMallocManaged((void**)&d_Filter_row_right_stream0, array_H * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_row_left_stream0, array_H * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_row_right_stream1, array_H * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_row_left_stream1, array_H * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_row_right_stream2, array_H * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_row_left_stream2, array_H * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_row_right_stream3, array_H * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_row_left_stream3, array_H * FILTER_RADIUS *sizeof(double));
    

    
    cudaCheckError();
    
    printf("start..\n");
    
    for(i=0; i < imageH/array_H; i++) {
        for(j=0; j < imageW/array_W; j += STREAMS) {
            
            hipMemcpyAsync(d_Input_stream0,&h_input_new[(i*imageH/array_H + j)*array_H*array_W],array_H*array_W*sizeof(double),hipMemcpyHostToDevice,stream0);
          
            hipMemcpyAsync(d_Filter_row_right_stream0,&h_filters_row_start[(i*(imageH/array_H-1) + j)*FILTER_RADIUS*array_H],array_H*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream0);
             
            
            if(j != 0) {                 
                hipMemcpyAsync(d_Filter_row_left_stream0,&h_filters_row_end[(i*(imageH/array_H-1) + j -1)*FILTER_RADIUS*array_H],array_H*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream0);
            }
            
            hipMemcpyAsync(d_Input_stream1,&h_input_new[(i*imageH/array_H + j + 1)*array_H*array_W],array_H*array_W*sizeof(double),hipMemcpyHostToDevice,stream1);
             
            hipMemcpyAsync(d_Filter_row_right_stream1,&h_filters_row_start[(i*(imageH/array_H-1) + j+1)*FILTER_RADIUS*array_H],array_H*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream1);
            
            hipMemcpyAsync(d_Filter_row_left_stream1,&h_filters_row_end[(i*(imageH/array_H-1) + j)*FILTER_RADIUS*array_H],array_H*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream1);
             
            hipMemcpyAsync(d_Input_stream2,&h_input_new[(i*imageH/array_H + j + 2)*array_H*array_W],array_H*array_W*sizeof(double),hipMemcpyHostToDevice,stream2);
             
            hipMemcpyAsync(d_Filter_row_right_stream2,&h_filters_row_start[(i*(imageH/array_H-1) + j+2)*FILTER_RADIUS*array_H],array_H*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream2);
              
            hipMemcpyAsync(d_Filter_row_left_stream2,&h_filters_row_end[(i*(imageH/array_H-1) + j + 1)*FILTER_RADIUS*array_H],array_H*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream2);
             
            hipMemcpyAsync(d_Input_stream3,&h_input_new[(i*imageH/array_H + j + 3)*array_H*array_W],array_H*array_W*sizeof(double),hipMemcpyHostToDevice,stream3);
             
            hipMemcpyAsync(d_Filter_row_left_stream3,&h_filters_row_end[(i*(imageH/array_H-1) + j + 2)*FILTER_RADIUS*array_H],array_H*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream3);
             
            if(j != (imageW/array_W) - STREAMS) {
                hipMemcpyAsync(d_Filter_row_right_stream3,&h_filters_row_start[(i*(imageH/array_H-1) + j+3)*FILTER_RADIUS*array_H],array_H*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream3);
            }
            
            if(j == 0) {
                convolutionRowGPU<<<blocks,threads,0,stream0>>>(d_Buffer_stream0,d_Input_stream0,d_Filter_row_right_stream0,d_Filter_row_right_stream0,array_W,array_H,1);
            }
            
            else {
                convolutionRowGPU<<<blocks,threads,0,stream0>>>(d_Buffer_stream0,d_Input_stream0,d_Filter_row_right_stream0,d_Filter_row_left_stream0,array_W,array_H,2);
            }
            
            convolutionRowGPU<<<blocks,threads,0,stream1>>>(d_Buffer_stream1,d_Input_stream1,d_Filter_row_right_stream1,d_Filter_row_left_stream1,array_W,array_H,2);
            
            convolutionRowGPU<<<blocks,threads,0,stream2>>>(d_Buffer_stream2,d_Input_stream2,d_Filter_row_right_stream2,d_Filter_row_left_stream2,array_W,array_H,2);
            
            if(j == (imageW/array_W) - STREAMS) {
                convolutionRowGPU<<<blocks,threads,0,stream3>>>(d_Buffer_stream3,d_Input_stream3,d_Filter_row_left_stream3,d_Filter_row_left_stream3,array_W,array_H,0);
            }
            
            else {
                convolutionRowGPU<<<blocks,threads,0,stream3>>>(d_Buffer_stream3,d_Input_stream3,d_Filter_row_right_stream3,d_Filter_row_left_stream3,array_W,array_H,2);
            }
            
            
            hipMemcpyAsync(&h_Buffer_new[(i*imageH/array_H + j)*array_H*array_W],d_Buffer_stream0,array_H * array_W * sizeof(double),hipMemcpyDeviceToHost,stream0);
            
            hipMemcpyAsync(&h_Buffer_new[(i*imageH/array_H + j+1)*array_H*array_W],d_Buffer_stream1,array_H * array_W * sizeof(double),hipMemcpyDeviceToHost,stream1);
            
            hipMemcpyAsync(&h_Buffer_new[(i*imageH/array_H + j+2)*array_H*array_W],d_Buffer_stream2,array_H * array_W * sizeof(double),hipMemcpyDeviceToHost,stream2);
            
            hipMemcpyAsync(&h_Buffer_new[(i*imageH/array_H + j+3)*array_H*array_W],d_Buffer_stream3,array_H * array_W * sizeof(double),hipMemcpyDeviceToHost,stream3);
            
        }
    }
    hipDeviceSynchronize();
    cudaCheckError();
    
    hipHostFree(h_input_new);
    hipHostFree(h_filters_row_start);
    hipHostFree(h_filters_row_end);
    
    cudaCheckError();
    
    hipFree(d_Input_stream0);
    hipFree(d_Input_stream1);
    hipFree(d_Input_stream2);
    hipFree(d_Input_stream3);
    
    hipFree(d_Filter_row_right_stream0);
    hipFree(d_Filter_row_right_stream1);
    hipFree(d_Filter_row_right_stream2);
    hipFree(d_Filter_row_right_stream3);
    
    hipFree(d_Filter_row_left_stream0);
    hipFree(d_Filter_row_left_stream1);
    hipFree(d_Filter_row_left_stream2);
    hipFree(d_Filter_row_left_stream3);
    
    cudaCheckError();
    
    
    hipHostMalloc((void**)&h_OutputGPU_new, imageW * imageH * sizeof(double), hipHostMallocDefault);
    cudaCheckError();
    hipHostMalloc((void**)& h_filters_col_up,(imageH/array_H-1) * imageW/array_W * array_W * FILTER_RADIUS *sizeof(double), hipHostMallocDefault);
    cudaCheckError();
    hipHostMalloc((void**)&  h_filters_col_down,(imageH/array_H-1) * imageW/array_W * array_W * FILTER_RADIUS *sizeof(double), hipHostMallocDefault);
    cudaCheckError();
    
    hipMallocManaged((void**)&d_Filter_col_up_stream0, array_W * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_col_up_stream1, array_W * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_col_up_stream2, array_W * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_col_up_stream3, array_W * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    
    hipMallocManaged((void**)&d_Filter_col_down_steam0, array_W * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_col_down_steam1, array_W * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_col_down_steam2, array_W * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_Filter_col_down_steam3, array_W * FILTER_RADIUS *sizeof(double));
    cudaCheckError();
    
    hipMallocManaged((void**)&d_OutputGPU_stream0,array_H * array_W * sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_OutputGPU_stream1,array_H * array_W * sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_OutputGPU_stream2,array_H * array_W * sizeof(double));
    cudaCheckError();
    hipMallocManaged((void**)&d_OutputGPU_stream3,array_H * array_W * sizeof(double));
    cudaCheckError();
    
    for(i=0; i < imageH/array_H; i++) {
        for(j=0; j < imageW/array_W; j++) {
            for(y=0; y < array_H; y++) {
                for(x=0; x < array_W;x++) {
                    
                    if(i != 0 && y < FILTER_RADIUS) {
                        h_filters_col_up[x+ y*array_W+((i-1)*(imageH/array_H)+j)*FILTER_RADIUS*array_W] = h_Buffer_new[y*array_W + x + (i*imageH/array_H + j)*array_H*array_W];
                    }
                    
                    if(i != imageH/array_H-1 && y + FILTER_RADIUS > array_H-1) {
                        h_filters_col_down[x + (y-array_H+FILTER_RADIUS)*array_W + (i*(imageH/array_H)+j)*FILTER_RADIUS*array_W] =  h_Buffer_new[y*array_W + x + (i*imageH/array_H + j)*array_H*array_W];
                    }
                    
                }
            }
        }
    }
    
    

    for(i=0; i < imageH/array_H; i += STREAMS) {
        for(j=0; j < imageW/array_W; j++ ) {
            
            hipMemcpyAsync(d_Buffer_stream0,&h_Buffer_new[(i*imageH/array_H + j)*array_H*array_W],array_H*array_W*sizeof(double),hipMemcpyHostToDevice,stream0);
            
            hipMemcpyAsync(d_Filter_col_down_steam0,&h_filters_col_up[(i*(imageH/array_H) + j)*FILTER_RADIUS*array_W],array_W*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream0);
            
            if(i != 0) {
                hipMemcpyAsync(d_Filter_col_up_stream0,&h_filters_col_down[((i-1)*(imageH/array_H) + j)*FILTER_RADIUS*array_W],array_W*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream0);
            }
            
            hipMemcpyAsync(d_Buffer_stream1,&h_Buffer_new[((i+1)*imageH/array_H + j)*array_H*array_W],array_H*array_W*sizeof(double),hipMemcpyHostToDevice,stream1);
            
            hipMemcpyAsync(d_Filter_col_down_steam1,&h_filters_col_up[((i+1)*(imageH/array_H) + j)*FILTER_RADIUS*array_W],array_W*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream1);
            
            hipMemcpyAsync(d_Filter_col_up_stream1,&h_filters_col_down[(i*(imageH/array_H) + j)*FILTER_RADIUS*array_W],array_W*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream1);
            
            hipMemcpyAsync(d_Buffer_stream2,&h_Buffer_new[((i+2)*imageH/array_H + j)*array_H*array_W],array_H*array_W*sizeof(double),hipMemcpyHostToDevice,stream2);
            
            hipMemcpyAsync(d_Filter_col_down_steam2,&h_filters_col_up[((i+2)*(imageH/array_H) + j)*FILTER_RADIUS*array_W],array_W*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream2);
            
            hipMemcpyAsync(d_Filter_col_up_stream2,&h_filters_col_down[((i+1)*(imageH/array_H) + j)*FILTER_RADIUS*array_W],array_W*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream2);
            
            hipMemcpyAsync(d_Buffer_stream3,&h_Buffer_new[((i+3)*imageH/array_H + j)*array_H*array_W],array_H*array_W*sizeof(double),hipMemcpyHostToDevice,stream3);
            
            hipMemcpyAsync(d_Filter_col_up_stream3,&h_filters_col_down[((i+2)*(imageH/array_H) + j)*FILTER_RADIUS*array_W],array_W*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream3);
            
            if(i != imageH/array_H - STREAMS) {
                hipMemcpyAsync(d_Filter_col_down_steam3,&h_filters_col_up[((i+3)*(imageH/array_H) + j)*FILTER_RADIUS*array_W],array_W*FILTER_RADIUS*sizeof(double),hipMemcpyHostToDevice,stream3);
            }
                
            
            
            if(i == 0) {
                convolutionColumnGPU<<<blocks,threads,0,stream0>>>(d_OutputGPU_stream0,d_Buffer_stream0,d_Filter_col_down_steam0,d_Filter_col_down_steam0,array_W,array_H,1);
            }
            else{
                convolutionColumnGPU<<<blocks,threads,0,stream0>>>(d_OutputGPU_stream0,d_Buffer_stream0,d_Filter_col_up_stream0,d_Filter_col_down_steam0,array_W,array_H,2);
            }
            
            convolutionColumnGPU<<<blocks,threads,0,stream1>>>(d_OutputGPU_stream1,d_Buffer_stream1,d_Filter_col_up_stream1,d_Filter_col_down_steam1,array_W,array_H,2);
                
            convolutionColumnGPU<<<blocks,threads,0,stream2>>>(d_OutputGPU_stream2,d_Buffer_stream2,d_Filter_col_up_stream2,d_Filter_col_down_steam2,array_W,array_H,2);
            
            
            if(i == imageH/array_H -1) {
                convolutionColumnGPU<<<blocks,threads,0,stream3>>>(d_OutputGPU_stream3,d_Buffer_stream3,d_Filter_col_up_stream3,d_Filter_col_up_stream3,array_W,array_H,0);
            }
            else{
                convolutionColumnGPU<<<blocks,threads,0,stream3>>>(d_OutputGPU_stream3,d_Buffer_stream3,d_Filter_col_up_stream3,d_Filter_col_down_steam3,array_W,array_H,2);
            }
            
            hipMemcpyAsync(&h_OutputGPU_new[(i*imageH/array_H + j)*array_H*array_W],d_OutputGPU_stream0,array_H * array_W * sizeof(double),hipMemcpyDeviceToHost,stream0);
            
            hipMemcpyAsync(&h_OutputGPU_new[((i+1)*imageH/array_H + j)*array_H*array_W],d_OutputGPU_stream1,array_H * array_W * sizeof(double),hipMemcpyDeviceToHost,stream1);
            
            hipMemcpyAsync(&h_OutputGPU_new[((i+2)*imageH/array_H + j)*array_H*array_W],d_OutputGPU_stream2,array_H * array_W * sizeof(double),hipMemcpyDeviceToHost,stream2);
            
            hipMemcpyAsync(&h_OutputGPU_new[((i+3)*imageH/array_H + j)*array_H*array_W],d_OutputGPU_stream3,array_H * array_W * sizeof(double),hipMemcpyDeviceToHost,stream3);
           
            
        }
    }
    
    
    hipDeviceSynchronize();
    cudaCheckError();
    
    hipFree(d_OutputGPU_stream0);
    hipFree(d_OutputGPU_stream1);
    hipFree(d_OutputGPU_stream2);
    hipFree(d_OutputGPU_stream3);
    
    hipFree(d_Buffer_stream0);
    hipFree(d_Buffer_stream1);
    hipFree(d_Buffer_stream2);
    hipFree(d_Buffer_stream3);
    
    hipFree(d_Filter_col_down_steam0);
    hipFree(d_Filter_col_down_steam1);
    hipFree(d_Filter_col_down_steam2);
    hipFree(d_Filter_col_down_steam3);
    
    hipFree(d_Filter_col_up_stream0);
    hipFree(d_Filter_col_up_stream1);
    hipFree(d_Filter_col_up_stream2);
    hipFree(d_Filter_col_up_stream3);
    
    
    for(i=0; i < imageH/array_H; i++) {
        for(j=0; j < imageW/array_W; j++) {
            for(y=0; y < array_H; y++) {
                for(x=0; x < array_W;x++) {
                    h_OutputGPU[(y+i*array_H)*imageW + x + j*array_W] = h_OutputGPU_new[y*array_W + x + (i*imageH/array_H + j)*array_H*array_W];
                }
            }
        }
    }
    
        
    hipHostFree(h_OutputGPU_new);
    hipHostFree(h_Buffer_new);
    hipHostFree(h_filters_col_down);
    hipHostFree(h_filters_col_up);
    
    
#ifdef _HOST    
    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    
    start = clock();
    
    convolutionRowCPU(h_Buffer_debug, h_Input, h_Filter, imageW, imageH); // convolution kata grammes
    
    convolutionColumnCPU(h_OutputCPU, h_Buffer_debug, h_Filter, imageW, imageH); // convolution kata sthles
    
    end = clock();
    
    timing = ((double) (end - start)) / CLOCKS_PER_SEC;
    
    printf("CPU computation : COMPLETED in time:%10.5f\n",timing);

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas 
    
    
    printf("\nCPU computations == GPU computation?\n");
    for (i = 0; i < imageW * imageH; i++) {
        if(h_OutputGPU[i] > h_OutputCPU[i] + accuracy || h_OutputGPU[i] < h_OutputCPU[i] - accuracy){
            printf("CPU computations == GPU computation : FALSE line:%d difrence:%f \nExitting program...\n GPU: %lf \n",i,h_OutputGPU[i]-h_OutputCPU[i],h_OutputGPU[i]);
           
            
            cudaDeviceReset();
            return(1);
        }
            
    }
    printf("CPU computations == GPU computation : TRUE \nExitting program after Memmory Free...\n");
#endif

    hipDeviceReset();
    return 0;
}
